#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

__global__ void spmvCSRKernel(float *out, int *matCols, int *matRows,
                              float *matData, float *vec, int dim) {
    //@@ insert spmv kernel for csr format
}

__global__ void spmvJDSKernel(float *out, int *matColStart, int *matCols,
                              int *matRowPerm, int *matRows, float *matData,
                              float *vec, int dim) {
    //@@ insert spmv kernel for jds format
}

static void spmvCSR(float *out, int *matCols, int *matRows, float *matData,
                    float *vec, int dim) {

    //@@ invoke spmv kernel for csr format
}

static void spmvJDS(float *out, int *matColStart, int *matCols, int *matRowPerm,
                    int *matRows, float *matData, float *vec, int dim) {

    //@@ invoke spmv kernel for jds format
}

int main(int argc, char **argv) {
  wbArg_t args;
  bool usingJDSQ;
  int *hostCSRCols;
  int *hostCSRRows;
  float *hostCSRData;
  int *hostJDSColStart;
  int *hostJDSCols;
  int *hostJDSRowPerm;
  int *hostJDSRows;
  float *hostJDSData;
  float *hostVector;
  float *hostOutput;
  int *deviceCSRCols;
  int *deviceCSRRows;
  float *deviceCSRData;
  int *deviceJDSColStart;
  int *deviceJDSCols;
  int *deviceJDSRowPerm;
  int *deviceJDSRows;
  float *deviceJDSData;
  float *deviceVector;
  float *deviceOutput;
  int dim, ncols, nrows, ndata;
  int maxRowNNZ;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  usingJDSQ = wbImport_flag(wbArg_getInputFile(args, 0)) == 1;
  hostCSRCols = (int *)wbImport(wbArg_getInputFile(args, 1), &ncols, "Integer");
  hostCSRRows = (int *)wbImport(wbArg_getInputFile(args, 2), &nrows, "Integer");
  hostCSRData = (float *)wbImport(wbArg_getInputFile(args, 3), &ndata, "Real");
  hostVector = (float *)wbImport(wbArg_getInputFile(args, 4), &dim, "Real");

  hostOutput = (float *)malloc(sizeof(float) * dim);

  wbTime_stop(Generic, "Importing data and creating memory on host");

  if (usingJDSQ) {
    CSRToJDS(dim, hostCSRRows, hostCSRCols, hostCSRData, &hostJDSRowPerm,
             &hostJDSRows, &hostJDSColStart, &hostJDSCols, &hostJDSData);
    maxRowNNZ = hostJDSRows[0];
  }

  wbTime_start(GPU, "Allocating GPU memory.");
  if (usingJDSQ) {
    hipMalloc((void **)&deviceJDSColStart, sizeof(int) * maxRowNNZ);
    hipMalloc((void **)&deviceJDSCols, sizeof(int) * ndata);
    hipMalloc((void **)&deviceJDSRowPerm, sizeof(int) * dim);
    hipMalloc((void **)&deviceJDSRows, sizeof(int) * dim);
    hipMalloc((void **)&deviceJDSData, sizeof(float) * ndata);
  } else {
    hipMalloc((void **)&deviceCSRCols, sizeof(int) * ncols);
    hipMalloc((void **)&deviceCSRRows, sizeof(int) * nrows);
    hipMalloc((void **)&deviceCSRData, sizeof(float) * ndata);
  }
  hipMalloc((void **)&deviceVector, sizeof(float) * dim);
  hipMalloc((void **)&deviceOutput, sizeof(float) * dim);
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  if (usingJDSQ) {
    hipMemcpy(deviceJDSColStart, hostJDSColStart, sizeof(int) * maxRowNNZ,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSCols, hostJDSCols, sizeof(int) * ndata,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSRowPerm, hostJDSRowPerm, sizeof(int) * dim,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSRows, hostJDSRows, sizeof(int) * dim,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceJDSData, hostJDSData, sizeof(float) * ndata,
               hipMemcpyHostToDevice);
  } else {
    hipMemcpy(deviceCSRCols, hostCSRCols, sizeof(int) * ncols,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceCSRRows, hostCSRRows, sizeof(int) * nrows,
               hipMemcpyHostToDevice);
    hipMemcpy(deviceCSRData, hostCSRData, sizeof(float) * ndata,
               hipMemcpyHostToDevice);
  }
  hipMemcpy(deviceVector, hostVector, sizeof(float) * dim,
             hipMemcpyHostToDevice);
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  wbTime_start(Compute, "Performing CUDA computation");
  if (usingJDSQ) {
    spmvJDS(deviceOutput, deviceJDSColStart, deviceJDSCols, deviceJDSRowPerm,
            deviceJDSRows, deviceJDSData, deviceVector, dim);
  } else {
    spmvCSR(deviceOutput, deviceCSRCols, deviceCSRRows, deviceCSRData,
            deviceVector, dim);
  }
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing CUDA computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  hipMemcpy(hostOutput, deviceOutput, sizeof(float) * dim,
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(hostCSRCols);
  hipFree(hostCSRRows);
  hipFree(hostCSRData);
  hipFree(hostVector);
  hipFree(hostOutput);
  if (usingJDSQ) {
    hipFree(hostJDSColStart);
    hipFree(hostJDSCols);
    hipFree(hostJDSRowPerm);
    hipFree(hostJDSRows);
    hipFree(hostJDSData);
  }
  wbTime_stop(GPU, "Freeing GPU Memory");

  wbSolution(args, hostOutput, dim);

  free(hostCSRCols);
  free(hostCSRRows);
  free(hostCSRData);
  free(hostVector);
  free(hostOutput);
  if (usingJDSQ) {
    free(hostJDSColStart);
    free(hostJDSCols);
    free(hostJDSRowPerm);
    free(hostJDSRows);
    free(hostJDSData);
  }

  return 0;
}
