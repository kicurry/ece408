#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

__host__ __device__ int outInvariant(int inValue) {
  return inValue * inValue;
}

__host__ __device__ int outDependent(int value, int inIdx, int outIdx) {
  if (inIdx == outIdx) {
    return 2 * value;
  } else if (inIdx > outIdx) {
    return value / (inIdx - outIdx);
  } else {
    return value / (outIdx - inIdx);
  }
}

__global__ void s2g_gpu_scatter_kernel(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
}

__global__ void s2g_gpu_gather_kernel(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
}

static void s2g_cpu_scatter(int *in, int *out, int len) {
  for (int inIdx = 0; inIdx < len; ++inIdx) {
    int intermediate = outInvariant(in[inIdx]);
    for (int outIdx = 0; outIdx < len; ++outIdx) {
      out[outIdx] += outDependent(intermediate, inIdx, outIdx);
    }
  }
}

static void s2g_cpu_gather(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
}

static void s2g_gpu_scatter(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
}

static void s2g_gpu_gather(int *in, int *out, int len) {
  //@@ INSERT CODE HERE
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  int *hostInput;
  int *hostOutput;
  int *deviceInput;
  int *deviceOutput;
  size_t byteCount;

  args = wbArg_read(argc, argv);

  wbTime_start(Generic, "Importing data and creating memory on host");
  hostInput =
      ( int * )wbImport(wbArg_getInputFile(args, 0), &inputLength, "Integer");
  hostOutput = ( int * )malloc(inputLength * sizeof(int));
  wbTime_stop(Generic, "Importing data and creating memory on host");

  wbLog(TRACE, "The input length is ", inputLength);
  byteCount = inputLength * sizeof(int);

  wbTime_start(GPU, "Allocating GPU memory.");
  wbCheck(hipMalloc(( void ** )&deviceInput, byteCount));
  wbCheck(hipMalloc(( void ** )&deviceOutput, byteCount));
  wbTime_stop(GPU, "Allocating GPU memory.");

  wbTime_start(GPU, "Copying input memory to the GPU.");
  wbCheck(
      hipMemcpy(deviceInput, hostInput, byteCount, hipMemcpyHostToDevice));
  wbCheck(hipMemset(deviceOutput, 0, byteCount));
  wbTime_stop(GPU, "Copying input memory to the GPU.");

  //////////////////////////////////////////
  // CPU Scatter Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing CPU Scatter computation");
  s2g_cpu_scatter(hostInput, hostOutput, inputLength);
  wbTime_stop(Compute, "Performing CPU Scatter computation");
  wbSolution(args, hostOutput, inputLength);
  memset(hostOutput, 0, byteCount);

  //////////////////////////////////////////
  // GPU Scatter Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Scatter computation");
  s2g_gpu_scatter(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Scatter computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(
      hipMemcpy(hostOutput, deviceOutput, byteCount, hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);
  wbCheck(hipMemset(deviceOutput, 0, byteCount));

  //////////////////////////////////////////
  // CPU Gather Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing CPU Gather computation");
  s2g_cpu_gather(hostInput, hostOutput, inputLength);
  wbTime_stop(Compute, "Performing CPU Gather computation");
  wbSolution(args, hostOutput, inputLength);
  memset(hostOutput, 0, byteCount);

  //////////////////////////////////////////
  // GPU Gather Computation
  //////////////////////////////////////////
  wbTime_start(Compute, "Performing GPU Gather computation");
  s2g_gpu_gather(deviceInput, deviceOutput, inputLength);
  hipDeviceSynchronize();
  wbTime_stop(Compute, "Performing GPU Gather computation");

  wbTime_start(Copy, "Copying output memory to the CPU");
  wbCheck(
      hipMemcpy(hostOutput, deviceOutput, byteCount, hipMemcpyDeviceToHost));
  wbTime_stop(Copy, "Copying output memory to the CPU");

  wbSolution(args, hostOutput, inputLength);
  wbCheck(hipMemset(deviceOutput, 0, byteCount));

  wbTime_start(GPU, "Freeing GPU Memory");
  hipFree(deviceInput);
  hipFree(deviceOutput);
  wbTime_stop(GPU, "Freeing GPU Memory");

  free(hostInput);
  free(hostOutput);

  return 0;
}
